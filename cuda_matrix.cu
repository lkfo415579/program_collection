//matrix_add.cu
//这个文件不推荐直接拷贝，最好一行一行地抄下来

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024  //这里定义了矩阵的阶级，这里用一个32x32的方形矩阵做例子
//想要自己Debug的时候，遇到问题，可以先把N的值设置小一些，比如4
#define THREAD_NUM 32
#define BLOCK_NUM 32
/*
 * 本次的核函数，三个参数分别是两个NxN的输入矩阵和一个NxN的输出矩阵
 */
__global__ void matrix_add(const int a[][N], const int b[][N], int c[][N]) {
    int idx = threadIdx.x;
    int idy = threadIdx.y;
    int bidx = blockIdx.x;
    int bidy = blockIdx.y;
    int realx = THREAD_NUM * bidx + idx;
    int realy = THREAD_NUM * bidy + idy;
    //printf("idx[%d],idy[%d]\n", idx, idy );
    //printf("realx[%d],realy[%d],bidx[%d]\n", realx,realy,bidx );
    //printf("bidx[%d]\n", bidx );
    //int idy = threadIdx.y;
    //printf ("a[idx][idy]:%d,idx[%d],idy[%d]\n", a[idx][idy], idx, idy );
    //printf ("a:%d,realx[%d],realy[%d]\n", a[realx][realy], realx, realy );
    c[realx][realy] = a[realx][realy] + b[realx][realy];
}

int main(void) {

    int *h_a, *h_b, *h_c;
    int *dev_a, *dev_b, *dev_c;

    //这里把一个block的里面的线程排列定义成二维的，这样会有两个维度的索引值，x和y
    dim3 threads_in_block (THREAD_NUM, THREAD_NUM);
    //err这个值是用来检查cuda的函数是否正常运行的
    hipError_t err = hipSuccess;

    h_a = (int *)malloc(sizeof(int) * N * N);
    h_b = (int *)malloc(sizeof(int) * N * N);
    h_c = (int *)malloc(sizeof(int) * N * N);

    if (h_a == NULL || h_b == NULL || h_c == NULL) {
        fprintf(stderr, "Malloc() failed.\n");
        return -1;
    }

    err = hipMalloc((void **)&dev_a, sizeof(int) * N * N);
    if (err != hipSuccess) {
        fprintf(stderr, "cudaMalloc() failed.\n");
        return -1;
    }
    err = hipMalloc((void **)&dev_b, sizeof(int) * N * N);
    if (err != hipSuccess) {
        fprintf(stderr, "cudaMalloc() failed.\n");
        return -1;
    }
    err = hipMalloc((void **)&dev_c, sizeof(int) * N * N);
    if (err != hipSuccess) {
        fprintf(stderr, "cudaMalloc() failed.\n");
        return -1;
    }

    for (int i = 0; i < N * N; i++) {
        h_a[i] = 2 * i + 1;
        h_b[i] = -1 * i + 5;
        //printf("a[%d]%d ", i, h_a[i] );
        //if (i % N == 0 && i != 0)
        //  printf("\n");
    }
    printf("\n--------------\n");
    //for (int i = 0; i < N * N; i++) {
      //printf("b[%d]%d ", i, h_b[i] );
      //if (i % N == 0 && i != 0)
        //printf("\n");
    //}
    //printf("\n");

    err = hipMemcpy(dev_a, h_a, sizeof(int) * N * N, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "cudaMemcpy() failed.\n");
        return -1;
    }
    err = hipMemcpy(dev_b, h_b, sizeof(int) * N * N, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "cudaMemcpy() failed.\n");
        return -1;
    }

    matrix_add<<<threads_in_block, threads_in_block>>>((int (*)[N])dev_a, (int (*)[N])dev_b, (int (*)[N])dev_c);

    err = hipMemcpy(h_c, dev_c, sizeof(int) * N * N, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "cudaMemcpy() failed.\n");
        return -1;
    }
    printf("done2.\n");
    for (int i = 0; i < N * N; i++) {
        if (h_a[i] + h_b[i] != h_c[i]) {
            fprintf(stderr, "a[%d]%d + b[%d]%d != c[%d]%d.\n", i, h_a[i], i, h_b[i], i, h_c[i]);
            return -1;
        }
    }
    for (int i = 0; i < N*N; i++) {
        printf("c[%d]%d ", i, h_c[i] );
        if (i % N == 0 && i != 0)
          printf("\n");
        if (i > 20)
          break;
    }
    printf("\nsize of c[]:%d\n",  N * N );
    //printf("\n");
    printf("done.\n");
    return 0;
}
